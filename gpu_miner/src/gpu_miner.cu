#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__global__ void findNonce(BYTE* block_content, BYTE* block_hash, BYTE* difficulty_hash, int* num_nonces_each_thread, uint64_t* nonce, int* nonce_found_flag) {
	int my_tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t starting_nonce = 1 + my_tid * (*num_nonces_each_thread);

	int len_initial_block_content = d_strlen((const char*) block_content);

	char *nonce_str;
	hipMalloc((void**) &nonce_str, NONCE_SIZE);

	int len_nonce;

	BYTE local_block_hash[BLOCK_SIZE];
	BYTE local_block_content[BLOCK_SIZE];
	d_strcpy((char*) local_block_content, (const char*) block_content);

	for (uint64_t i = starting_nonce; i < starting_nonce + *num_nonces_each_thread && i <= MAX_NONCE; ++i) {
		if (*nonce_found_flag) {
			hipFree((void*) nonce_str);
			return;
		}

		len_nonce = intToString(i, nonce_str);
		
		d_strcpy((char*) local_block_content + len_initial_block_content, (const char*) nonce_str);
		apply_sha256(local_block_content, len_initial_block_content + len_nonce, local_block_hash, 1);

		if (compare_hashes(local_block_hash, difficulty_hash) <= 0 && !(*nonce_found_flag)) {
			atomicExch(nonce_found_flag, 1);
			d_strcpy((char*) block_hash, (const char*) local_block_hash);

			*nonce = i;
			hipFree((void*) nonce_str);
			return;
		}
	}
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];

	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	uint64_t *nonce = 0;
	hipMalloc((void**)&nonce, sizeof(uint64_t));

	BYTE* block_hash;
	hipMalloc((void**)&block_hash, BLOCK_SIZE);

	BYTE* block_content_device;
	hipMalloc((void**)&block_content_device, BLOCK_SIZE);
	hipMemcpy(block_content_device, block_content, current_length + 1, hipMemcpyHostToDevice);

	// get the number of nonces each thread will check
	int *num_nonces_each_thread;
	hipMalloc((void**) &num_nonces_each_thread, sizeof(int));

	int num_threads = 512;
	size_t blocks_no = 64;

	int num_nonces_aux;
	if ((int)(MAX_NONCE) % ((int)blocks_no * (int)num_threads) == 0) {
		num_nonces_aux = MAX_NONCE / (blocks_no * num_threads);
	} else {
		num_nonces_aux = MAX_NONCE / (blocks_no * num_threads);
		++num_nonces_aux;
	}

	hipMemcpy((void*) num_nonces_each_thread, (void*) &num_nonces_aux, sizeof(int), hipMemcpyHostToDevice);
	
	BYTE* difficulty_hash;
	hipMalloc((void**)&difficulty_hash, SHA256_HASH_SIZE);
	hipMemcpy(difficulty_hash, DIFFICULTY, SHA256_HASH_SIZE, hipMemcpyHostToDevice);

	int* nonce_found_flag;
	hipMalloc((void**) &nonce_found_flag, sizeof(int));
	hipMemset(nonce_found_flag, 0, sizeof(int));
	
	hipEvent_t start, stop;
	startTiming(&start, &stop);
	
	findNonce<<<blocks_no, num_threads>>>(block_content_device, block_hash, difficulty_hash, num_nonces_each_thread, nonce, nonce_found_flag);
	hipDeviceSynchronize();
	
	float seconds = stopTiming(&start, &stop);

	BYTE* block_hash_host = (BYTE*) malloc(BLOCK_SIZE);
	hipMemcpy(block_hash_host, block_hash, BLOCK_SIZE, hipMemcpyDeviceToHost);

	uint64_t nonce_host;
	hipMemcpy(&nonce_host, nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);

	// free GPU memory
	hipFree(block_hash);
	hipFree(block_content_device);
	hipFree(difficulty_hash);
	hipFree(nonce);
	hipFree(nonce_found_flag);
	hipFree(num_nonces_each_thread);

	printResult(block_hash_host, nonce_host, seconds);

	free(block_hash_host);

	return 0;
}
